
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#define DICE_MIN 1
#define DICE_MAX 3
#define NUM_CAMELS 5
#define FULL_MASK 0xffffffff

__global__ void setup_kernel(hiprandState *state) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hiprand_init((unsigned long long)clock() + idx, idx, 0, &state[idx]);
}

template <typename T>
__global__ void camel_up_sim(hiprandState *state, const int *positions,
                             const bool *remaining_dice, const int *stack,
                             T *results, const T local_runs) {
  int thread_idx = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + thread_idx;

  __shared__ T shared_results[NUM_CAMELS];

  if (idx < NUM_CAMELS) {
    shared_results[thread_idx] = 0;
  }
  __syncthreads();

  T thread_results[NUM_CAMELS] = {0};

  // Save the global variables in the local thread
  // so we can reuse them without having to re-read globally.
  int saved_local_positions[NUM_CAMELS];
  bool saved_local_dice[NUM_CAMELS];
  int saved_local_stack[NUM_CAMELS];

  for (int i = 0; i < NUM_CAMELS; i++) {
    saved_local_positions[i] = positions[i];
    saved_local_dice[i] = remaining_dice[i];
    saved_local_stack[i] = stack[i];
  }

  // Instantiate versions of this that can be used within the
  // simulation.
  int local_positions[NUM_CAMELS];
  bool local_dice[NUM_CAMELS];
  int local_stack[NUM_CAMELS];
  bool moved_camels[NUM_CAMELS] = {0, 0, 0, 0, 0};
  int dice_remaining;

  int camel_to_move;
  int roll;
  int camel_on_top;
  int winner;

  for (int r = 0; r < local_runs; r++) {
    // Begin one simulation
    dice_remaining = 0;
    
    #pragma unroll
    for (int i = 0; i < NUM_CAMELS; i++) {
      // reset local arrays back to saved initial state.
      local_positions[i] = saved_local_positions[i];
      local_dice[i] = saved_local_dice[i];
      local_stack[i] = saved_local_stack[i];

      if (local_dice[i] == 1) {
        dice_remaining++;
      }
    }

    while (dice_remaining > 0) {

      do {
        camel_to_move = hiprand(&state[idx]) % NUM_CAMELS;
      } while (!local_dice[camel_to_move]);

      roll = hiprand(&state[idx]) % DICE_MAX + 1;

      // move that camel and set its dice as rolled.
      local_positions[camel_to_move] += roll;
      local_dice[camel_to_move] = 0;

#pragma unroll
      for (int j = 0; j < NUM_CAMELS; j++) {
        moved_camels[j] = 0;
      }
      moved_camels[camel_to_move] = 1;

      camel_on_top = local_stack[camel_to_move];

      // Move anyone who is on top of the camel that's moving
      while (camel_on_top != -1) {
        local_positions[camel_on_top] += roll;
        moved_camels[camel_on_top] = 1;
        camel_on_top = local_stack[camel_on_top];
      }

#pragma unroll
      for (int i = 0; i < NUM_CAMELS; i++) {
        // If anyone was on the space the stack moved to, make that camel point
        // to the bottom of the new stack
        if ((i != camel_to_move) &&
            (local_positions[i] == local_positions[camel_to_move]) &&
            (local_stack[i] == -1) && (!moved_camels[i])) {
          local_stack[i] = camel_to_move;
        } else if ((local_stack[i] == camel_to_move) &&
                   (local_positions[i] < local_positions[camel_to_move])) {
          // If anyone pointed to camel_to_move and is on a previous space
          // then make them uncovered.
          local_stack[i] = -1;
        }
      }

      dice_remaining--;
    }

    winner = 0;
#pragma unroll
    for (int i = 1; i < NUM_CAMELS; i++) {
      if (local_positions[i] > local_positions[winner]) {
        winner = i;
      }
    }

    while (local_stack[winner] != -1) {
      winner = local_stack[winner];
    }

    thread_results[winner] += 1;
  }

#pragma unroll
  for (int i = 0; i < NUM_CAMELS; i++) {
    for (int offset = 16; offset > 0; offset /= 2) {
      thread_results[i] +=
          __shfl_down_sync(FULL_MASK, thread_results[i], offset);
    }

    // If it's the first thread in a warp - report the result.
    if (threadIdx.x % 32 == 0) {
      atomicAdd(&shared_results[i], thread_results[i]);
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
#pragma unroll
    for (int i = 0; i < NUM_CAMELS; i++) {
      atomicAdd(&results[i], shared_results[i]);
    }
  }
}

template <typename T> void printArray(T arr[], int size) {
  std::cout << "[";
  for (int i = 0; i < size; i++) {
    std::cout << arr[i];
    if (i < size - 1) {
      std::cout << (", ");
    }
  }
  std::cout << "]\n";
}

int main() {

  using T = unsigned long long int;

  std::cout << "Starting program..." << std::endl;
  constexpr int BLOCKS = 24 * 4; // Four per SM on the 4060
  constexpr int THREADS = 256;
  constexpr int RUNS_PER_THREAD = 100000;
  // Without casting one of these to unsigned long long int then this can
  // overflow integer multiplication and return something nonsensical.
  constexpr unsigned long long int N =
      static_cast<unsigned long long int>(BLOCKS) * THREADS * RUNS_PER_THREAD;

  std::cout << "N: " << std::to_string(N) << std::endl;

  std::cout << "Creating host variables..." << std::endl;
  int positions[NUM_CAMELS] = {0, 0, 0, 0, 0};
  bool remainingDice[NUM_CAMELS] = {1, 1, 1, 1, 1};
  int stack[NUM_CAMELS] = {1, 2, 3, 4, -1};
  T *results;
  results = (T *)malloc(NUM_CAMELS * sizeof(T));

  std::cout << "Creating device pointers..." << std::endl;
  int *d_positions;
  bool *d_remainingDice;
  int *d_stack;
  T *d_results;

  hiprandState *d_state;
  hipMalloc((void **)&d_state, BLOCKS * THREADS * sizeof(hiprandState));

  std::cout << "Setting up hiprand states..." << std::endl;
  setup_kernel<<<BLOCKS, THREADS>>>(d_state);

  std::cout << "Allocating memory on device..." << std::endl;
  hipMalloc((void **)&d_positions, NUM_CAMELS * sizeof(int));
  hipMalloc((void **)&d_results, NUM_CAMELS * sizeof(T));
  hipMalloc((void **)&d_remainingDice, NUM_CAMELS * sizeof(bool));
  hipMalloc((void **)&d_stack, NUM_CAMELS * sizeof(int));

  hipMemset(d_results, 0, NUM_CAMELS * sizeof(T));

  std::cout << "Copying to device..." << std::endl;
  hipMemcpy(d_positions, positions, NUM_CAMELS * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_remainingDice, remainingDice, NUM_CAMELS * sizeof(bool),
             hipMemcpyHostToDevice);
  hipMemcpy(d_stack, stack, NUM_CAMELS * sizeof(int), hipMemcpyHostToDevice);

  std::cout << "Starting sim..." << std::endl;
  camel_up_sim<T><<<BLOCKS, THREADS>>>(d_state, d_positions, d_remainingDice,
                                       d_stack, d_results, RUNS_PER_THREAD);

  hipDeviceSynchronize();

  std::cout << "Copying results back..." << std::endl;
  hipMemcpy(results, d_results, NUM_CAMELS * sizeof(T),
             hipMemcpyDeviceToHost);

  std::cout << "Results are:" << std::endl;
  printArray(results, NUM_CAMELS);

  float probs[NUM_CAMELS];
  constexpr float N_float = static_cast<float>(N);
  for (int i = 0; i < NUM_CAMELS; i++) {
    probs[i] = static_cast<float>(results[i]) / N_float;
  }

  std::cout << "Probabilities are..." << std::endl;
  printArray(probs, NUM_CAMELS);

  hipFree(d_positions);
  hipFree(d_results);
  hipFree(d_remainingDice);
  hipFree(d_state);
  hipFree(d_stack);

  free(results);
}